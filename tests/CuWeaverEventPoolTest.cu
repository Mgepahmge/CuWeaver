#ifdef __HIPCC__

#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <thread>
#include <chrono>
#include <cuweaver/EventPool.cuh>

namespace cuweaver {

class EventPoolTest : public ::testing::Test {
protected:
    void SetUp() override {
        // Initialize CUDA context for testing
        hipError_t err = hipSetDevice(0);
        ASSERT_EQ(err, hipSuccess) << "Failed to set CUDA device";
        
        // Create test pool with moderate size
        pool = std::make_unique<EventPool>(5);
    }

    void TearDown() override {
        pool.reset();
        hipDeviceReset();
    }

    std::unique_ptr<EventPool> pool;
};

// Basic functionality tests
TEST_F(EventPoolTest, ConstructorInitializesCorrectly) {
    // Test that constructor creates a pool with specified size
    auto testPool = std::make_unique<EventPool>(3);
    ASSERT_NE(testPool, nullptr);
}

TEST_F(EventPoolTest, AcquireSingleEvent) {
    // Test acquiring a single event from the pool
    cudaEvent& event = pool->acquire();
    
    // Verify the event is valid by checking its native handle
    hipEvent_t handle = event.nativeHandle();
    ASSERT_NE(handle, nullptr);
    
    // Test that the event can be used in CUDA operations
    hipError_t err = hipEventRecord(handle, 0);
    EXPECT_EQ(err, hipSuccess);
}

TEST_F(EventPoolTest, AcquireMultipleEvents) {
    // Test acquiring multiple events from the pool
    std::vector<std::reference_wrapper<cudaEvent>> events;
    
    // Acquire initial pool size events
    for (size_t i = 0; i < 5; ++i) {
        events.emplace_back(pool->acquire());
    }
    
    // Verify all events are unique
    std::set<hipEvent_t> handles;
    for (auto& eventRef : events) {
        hipEvent_t handle = eventRef.get().nativeHandle();
        ASSERT_NE(handle, nullptr);
        EXPECT_TRUE(handles.insert(handle).second) << "Duplicate event handle found";
    }
}

TEST_F(EventPoolTest, ReleaseValidEvent) {
    // Test releasing a valid event back to the pool
    cudaEvent& event = pool->acquire();
    hipEvent_t originalHandle = event.nativeHandle();
    
    // Use the event for some CUDA operation
    ASSERT_EQ(hipEventRecord(originalHandle, 0), hipSuccess);
    
    // Release the event
    bool result = pool->release(event);
    EXPECT_TRUE(result) << "Failed to release valid event";
}

TEST_F(EventPoolTest, ReleaseInvalidEvent) {
    // Test releasing an event that wasn't acquired from this pool
    cudaEvent externalEvent(cudaEventFlags::DisableTiming);
    
    // Try to release an external event
    bool result = pool->release(externalEvent);
    EXPECT_FALSE(result) << "Should fail to release external event";
}

TEST_F(EventPoolTest, AcquireReleaseAcquireCycle) {
    // Test the complete acquire-release-acquire cycle
    cudaEvent& event1 = pool->acquire();
    hipEvent_t handle1 = event1.nativeHandle();
    
    // Use the event
    ASSERT_EQ(hipEventRecord(handle1, 0), hipSuccess);
    ASSERT_EQ(hipEventSynchronize(handle1), hipSuccess);
    
    // Release the event
    ASSERT_TRUE(pool->release(event1));
    
    // Acquire another event (should potentially reuse the released one)
    cudaEvent& event2 = pool->acquire();
    hipEvent_t handle2 = event2.nativeHandle();
    ASSERT_NE(handle2, nullptr);
    
    // The handle might be the same (reused) or different (new allocation)
    // Both scenarios are valid depending on pool implementation
}

TEST_F(EventPoolTest, PoolExpansionBeyondInitialSize) {
    // Test that pool can expand beyond initial size
    std::vector<std::reference_wrapper<cudaEvent>> events;
    
    // Acquire more events than initial pool size
    for (size_t i = 0; i < 10; ++i) {
        events.emplace_back(pool->acquire());
    }
    
    // Verify all events are valid and unique
    std::set<hipEvent_t> handles;
    for (auto& eventRef : events) {
        hipEvent_t handle = eventRef.get().nativeHandle();
        ASSERT_NE(handle, nullptr);
        EXPECT_TRUE(handles.insert(handle).second);
    }
    
    EXPECT_EQ(handles.size(), 10);
}

TEST_F(EventPoolTest, MultipleReleasesSameEvent) {
    // Test releasing the same event multiple times
    cudaEvent& event = pool->acquire();
    
    // First release should succeed
    EXPECT_TRUE(pool->release(event));
    
    // Second release of same event should fail
    EXPECT_FALSE(pool->release(event));
}

// Performance and stress tests
TEST_F(EventPoolTest, HighFrequencyAcquireRelease) {
    // Test high-frequency acquire and release operations
    const int iterations = 1000;
    
    for (int i = 0; i < iterations; ++i) {
        cudaEvent& event = pool->acquire();
        hipEvent_t handle = event.nativeHandle();
        
        // Simulate some work
        ASSERT_EQ(hipEventRecord(handle, 0), hipSuccess);
        
        // Release immediately
        ASSERT_TRUE(pool->release(event));
    }
}

TEST_F(EventPoolTest, BatchAcquireAndRelease) {
    // Test acquiring a batch of events and then releasing them
    const size_t batchSize = 50;
    std::vector<std::reference_wrapper<cudaEvent>> events;
    
    // Acquire batch
    for (size_t i = 0; i < batchSize; ++i) {
        events.emplace_back(pool->acquire());
    }
    
    // Use all events
    for (auto& eventRef : events) {
        hipEvent_t handle = eventRef.get().nativeHandle();
        ASSERT_EQ(hipEventRecord(handle, 0), hipSuccess);
    }
    
    // Wait for all events
    for (auto& eventRef : events) {
        hipEvent_t handle = eventRef.get().nativeHandle();
        ASSERT_EQ(hipEventSynchronize(handle), hipSuccess);
    }
    
    // Release all events
    for (auto& eventRef : events) {
        EXPECT_TRUE(pool->release(eventRef.get()));
    }
}

// Real-world simulation tests
TEST_F(EventPoolTest, StreamSynchronizationSimulation) {
    // Simulate real-world stream synchronization scenario
    hipStream_t stream1, stream2;
    ASSERT_EQ(hipStreamCreate(&stream1), hipSuccess);
    ASSERT_EQ(hipStreamCreate(&stream2), hipSuccess);
    
    // Acquire events for synchronization
    cudaEvent& event1 = pool->acquire();
    cudaEvent& event2 = pool->acquire();
    
    hipEvent_t handle1 = event1.nativeHandle();
    hipEvent_t handle2 = event2.nativeHandle();
    
    // Record events on different streams
    ASSERT_EQ(hipEventRecord(handle1, stream1), hipSuccess);
    ASSERT_EQ(hipEventRecord(handle2, stream2), hipSuccess);
    
    // Synchronize streams using events
    ASSERT_EQ(hipStreamWaitEvent(stream2, handle1, 0), hipSuccess);
    ASSERT_EQ(hipStreamWaitEvent(stream1, handle2, 0), hipSuccess);
    
    // Synchronize and cleanup
    ASSERT_EQ(hipStreamSynchronize(stream1), hipSuccess);
    ASSERT_EQ(hipStreamSynchronize(stream2), hipSuccess);
    
    // Release events
    EXPECT_TRUE(pool->release(event1));
    EXPECT_TRUE(pool->release(event2));
    
    // Cleanup streams
    ASSERT_EQ(hipStreamDestroy(stream1), hipSuccess);
    ASSERT_EQ(hipStreamDestroy(stream2), hipSuccess);
}

TEST_F(EventPoolTest, ResourceExhaustionAndRecovery) {
    // Test behavior when exhausting and recovering pool resources
    std::vector<std::reference_wrapper<cudaEvent>> events;
    
    // Acquire many events to potentially exhaust initial pool
    for (size_t i = 0; i < 100; ++i) {
        events.emplace_back(pool->acquire());
    }
    
    // Release half of them
    for (size_t i = 0; i < 50; ++i) {
        EXPECT_TRUE(pool->release(events[i].get()));
    }
    
    // Acquire new events (should reuse released ones)
    std::vector<std::reference_wrapper<cudaEvent>> newEvents;
    for (size_t i = 0; i < 25; ++i) {
        newEvents.emplace_back(pool->acquire());
    }
    
    // Verify new events are valid
    for (auto& eventRef : newEvents) {
        hipEvent_t handle = eventRef.get().nativeHandle();
        ASSERT_NE(handle, nullptr);
        ASSERT_EQ(hipEventRecord(handle, 0), hipSuccess);
    }
    
    // Release remaining events
    for (size_t i = 50; i < 100; ++i) {
        EXPECT_TRUE(pool->release(events[i].get()));
    }
    for (auto& eventRef : newEvents) {
        EXPECT_TRUE(pool->release(eventRef.get()));
    }
}

// Edge cases and error handling
TEST_F(EventPoolTest, ZeroSizePool) {
    // Test creating a pool with zero initial size
    auto zeroPool = std::make_unique<EventPool>(0);
    
    // Should still be able to acquire events (pool should expand)
    cudaEvent& event = zeroPool->acquire();
    hipEvent_t handle = event.nativeHandle();
    ASSERT_NE(handle, nullptr);
    
    EXPECT_TRUE(zeroPool->release(event));
}

TEST_F(EventPoolTest, LargeBatchOperations) {
    // Test with larger batch sizes to stress test the implementation
    const size_t largeBatch = 500;
    std::vector<std::reference_wrapper<cudaEvent>> events;
    
    // Acquire large batch
    for (size_t i = 0; i < largeBatch; ++i) {
        events.emplace_back(pool->acquire());
    }
    
    // Verify all are unique and valid
    std::set<hipEvent_t> handles;
    for (auto& eventRef : events) {
        hipEvent_t handle = eventRef.get().nativeHandle();
        ASSERT_NE(handle, nullptr);
        EXPECT_TRUE(handles.insert(handle).second);
    }
    
    // Release in reverse order
    for (auto it = events.rbegin(); it != events.rend(); ++it) {
        EXPECT_TRUE(pool->release(it->get()));
    }
}

}

#endif // __HIPCC__