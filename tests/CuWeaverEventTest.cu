#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include "cuweaver/Event.cuh"

using cuweaver::cudaEvent;

namespace {

inline bool cudaAvailable() {
    int count = 0;
    auto st = hipGetDeviceCount(&count);
    return (st == hipSuccess) && (count > 0);
}

inline void recordAndSync(hipEvent_t ev) {
    ASSERT_NE(ev, nullptr);
    ASSERT_EQ(hipEventRecord(ev, /*stream=*/0), hipSuccess);
    ASSERT_EQ(hipEventSynchronize(ev), hipSuccess);
}

} // namespace

TEST(CuWeaverCudaEvent, DefaultConstructorCreatesValidEvent) {
#ifndef __HIPCC__
    GTEST_SKIP() << "Not compiled with CUDA (__HIPCC__ not defined).";
#endif
    if (!cudaAvailable()) GTEST_SKIP() << "No CUDA device available.";

    cudaEvent e; // 默认构造
    EXPECT_TRUE(e.isValid());
    EXPECT_NE(e.nativeHandle(), nullptr);
    EXPECT_EQ(e.getFlags(), static_cast<cudaEvent::cudaEventFlags_t>(cuweaver::cudaEventFlags::Default));

    recordAndSync(e.nativeHandle());
}

TEST(CuWeaverCudaEvent, EnumFlagsConstructorDisableTiming) {
#ifndef __HIPCC__
    GTEST_SKIP() << "Not compiled with CUDA (__HIPCC__ not defined).";
#endif
    if (!cudaAvailable()) GTEST_SKIP() << "No CUDA device available.";

    cudaEvent e{cuweaver::cudaEventFlags::DisableTiming};
    EXPECT_TRUE(e.isValid());
    EXPECT_NE(e.nativeHandle(), nullptr);
    EXPECT_EQ(e.getFlags(), static_cast<unsigned int>(cuweaver::cudaEventFlags::DisableTiming));

    recordAndSync(e.nativeHandle());
}

TEST(CuWeaverCudaEvent, RawFlagsConstructor) {
#ifndef __HIPCC__
    GTEST_SKIP() << "Not compiled with CUDA (__HIPCC__ not defined).";
#endif
    if (!cudaAvailable()) GTEST_SKIP() << "No CUDA device available.";

    unsigned int rawFlags = static_cast<unsigned int>(cuweaver::cudaEventFlags::BlockingSync) |
                            static_cast<unsigned int>(cuweaver::cudaEventFlags::DisableTiming);
    cudaEvent e{rawFlags};
    EXPECT_TRUE(e.isValid());
    EXPECT_NE(e.nativeHandle(), nullptr);
    EXPECT_EQ(e.getFlags(), rawFlags);

    recordAndSync(e.nativeHandle());
}

TEST(CuWeaverCudaEvent, AdoptExistingNativeHandleConstructor) {
#ifndef __HIPCC__
    GTEST_SKIP() << "Not compiled with CUDA (__HIPCC__ not defined).";
#endif
    if (!cudaAvailable()) GTEST_SKIP() << "No CUDA device available.";

    hipEvent_t raw = nullptr;
    unsigned int rawFlags = static_cast<unsigned int>(cuweaver::cudaEventFlags::DisableTiming);
    ASSERT_EQ(hipEventCreateWithFlags(&raw, rawFlags), hipSuccess);
    ASSERT_NE(raw, nullptr);

    {
        cudaEvent e{raw};
        EXPECT_TRUE(e.isValid());
        EXPECT_EQ(e.nativeHandle(), raw);

        EXPECT_EQ(e.getFlags(), static_cast<unsigned int>(cuweaver::cudaEventFlags::Default));

        recordAndSync(e.nativeHandle());
    }

}


TEST(CuWeaverCudaEvent, MoveConstructorTransfersOwnership) {
#ifndef __HIPCC__
    GTEST_SKIP() << "Not compiled with CUDA (__HIPCC__ not defined).";
#endif
    if (!cudaAvailable()) GTEST_SKIP() << "No CUDA device available.";

    cudaEvent e1;
    ASSERT_TRUE(e1.isValid());
    auto h = e1.nativeHandle();

    cudaEvent e2{std::move(e1)};
    EXPECT_FALSE(e1.isValid());
    EXPECT_TRUE(e2.isValid());
    EXPECT_EQ(e2.nativeHandle(), h);

    recordAndSync(e2.nativeHandle());
}

TEST(CuWeaverCudaEvent, MoveAssignmentTransfersOwnership) {
#ifndef __HIPCC__
    GTEST_SKIP() << "Not compiled with CUDA (__HIPCC__ not defined).";
#endif
    if (!cudaAvailable()) GTEST_SKIP() << "No CUDA device available.";

    cudaEvent src;
    ASSERT_TRUE(src.isValid());
    auto hsrc = src.nativeHandle();

    cudaEvent dst{cuweaver::cudaEventFlags::DisableTiming};
    ASSERT_TRUE(dst.isValid());

    dst = std::move(src);
    EXPECT_FALSE(src.isValid());
    EXPECT_TRUE(dst.isValid());
    EXPECT_EQ(dst.nativeHandle(), hsrc);

    recordAndSync(dst.nativeHandle());
}

// -------------------- reset / isValid --------------------

TEST(CuWeaverCudaEvent, ResetToNewHandleAndNullptr) {
#ifndef __HIPCC__
    GTEST_SKIP() << "Not compiled with CUDA (__HIPCC__ not defined).";
#endif
    if (!cudaAvailable()) GTEST_SKIP() << "No CUDA device available.";

    cudaEvent e;
    EXPECT_TRUE(e.isValid());
    auto old = e.nativeHandle();

    hipEvent_t nraw = nullptr;
    ASSERT_EQ(hipEventCreate(&nraw), hipSuccess);
    ASSERT_NE(nraw, nullptr);

    e.reset(nraw);
    EXPECT_TRUE(e.isValid());
    EXPECT_EQ(e.nativeHandle(), nraw);
    recordAndSync(e.nativeHandle());

    e.reset(nullptr);
    EXPECT_FALSE(e.isValid());
    EXPECT_EQ(e.nativeHandle(), static_cast<hipEvent_t>(nullptr));

    (void)old;
}

TEST(CuWeaverCudaEvent, NativeHandleConstCorrectness) {
#ifndef __HIPCC__
    GTEST_SKIP() << "Not compiled with CUDA (__HIPCC__ not defined).";
#endif
    if (!cudaAvailable()) GTEST_SKIP() << "No CUDA device available.";

    const cudaEvent e;
    EXPECT_TRUE(e.isValid());
    EXPECT_NE(e.nativeHandle(), nullptr);
}

