#include <cuweaver_utils/Error.cuh>

namespace cuweaver {
    const char* cudaErrorCategoryImpl::name() const noexcept {
        return "cuda";
    }

    std::string cudaErrorCategoryImpl::message(int ev) const {
        auto e = static_cast<hipError_t>(ev);
        if (const char* s = hipGetErrorString(e)) {
            return s;
        }
        return "Unknown CUDA error";
    }

    hipError_t::hipError_t(hipError_t e, cudaErrorContext ctx) : std::system_error(makeErrorCode(e),
                                                                    buildWhat(ctx)),
                                                                code_(e),
                                                                ctx_(std::move(ctx)) {
    }

    hipError_t hipError_t::codeNative() const noexcept {
        return code_;
    }

    const cudaErrorContext& hipError_t::context() const noexcept {
        return ctx_;
    }

    std::string hipError_t::buildWhat(const cudaErrorContext& c) {
        std::string s = c.op;
        if (!c.detail.empty()) {
            s += " | ";
            s += c.detail;
        }
        s += " @ ";
        s += c.loc.file;
        s += ":";
        s += std::to_string(c.loc.line);
        return s;
    }
}
