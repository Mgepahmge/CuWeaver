#include <cuweaver/cudaDevice.cuh>

namespace cuweaver {
    cudaDevice::cudaDevice(const int deviceId) : deviceId(deviceId) {
    }

    detail::cudaDeviceProperties cudaDevice::getProp() const {
        hipDeviceProp_t prop{};
        CUW_THROW_IF_ERROR(cudaGetDeviceProperties_v2(&prop, deviceId));
        return detail::cudaDeviceProperties(prop);
    }

    void cudaDevice::setDevice() const {
        int current = 0;
        CUW_THROW_IF_ERROR(hipGetDevice(&current));
        if (current != deviceId) {
            CUW_THROW_IF_ERROR(hipSetDevice(deviceId));
        }
    }

    void cudaDevice::setFlags(cudaDeviceFlags flags) const {
        switchContext([flags] {
            CUW_THROW_IF_ERROR(hipSetDeviceFlags(static_cast<unsigned int>(flags)));
        });
    }

    cudaDeviceFlags cudaDevice::getFlags() const {
        unsigned flags = 0;
        switchContext([&flags] {
            unsigned int flags_ = 0;
            CUW_THROW_IF_ERROR(hipGetDeviceFlags(&flags_));
            flags = flags_;
        });
        return static_cast<cudaDeviceFlags>(flags);
    }

    void cudaDevice::synchronize() const {
        switchContext([] {
            CUW_THROW_IF_ERROR(hipDeviceSynchronize());
        });
    }

    void cudaDevice::reset() const {
        switchContext([] {
            CUW_THROW_IF_ERROR(hipDeviceReset());
        });
    }

    std::string cudaDevice::getPCIBusId(const unsigned int len) const {
        auto result = new char[len];
        CUW_THROW_IF_ERROR(hipDeviceGetPCIBusId(result, len, deviceId));
        std::string str(result);
        delete[] result;
        return str;
    }
}
