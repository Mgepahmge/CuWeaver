#include <cuweaver/EventStreamOps.cuh>

#include "cuweaver_utils/ErrorCheck.cuh"

namespace cuweaver {
    float eventElapsedTime(const cudaEvent& start, const cudaEvent& end) {
        float ms;
        CUW_THROW_IF_ERROR(hipEventElapsedTime(&ms, start.nativeHandle(), end.nativeHandle()));
        return ms;
    }

    bool eventQuery(const cudaEvent& event) {
        try {
            CUW_THROW_IF_ERROR(hipEventQuery(event.nativeHandle()));
            return true;
        }
        catch (const hipError_t& e) {
            if (e.codeNative() == hipErrorNotReady) {
                return false;
            }
            throw;
        }
    }

    void eventRecord(const cudaEvent& event, const cudaStream& stream) {
        CUW_THROW_IF_ERROR(hipEventRecord(event.nativeHandle(), stream.nativeHandle()));
    }

    void eventRecordWithFlags(const cudaEvent& event, const cudaStream& stream, cudaEventRecordFlags flags) {
        CUW_THROW_IF_ERROR(
            hipEventRecordWithFlags(event.nativeHandle(), stream.nativeHandle(), static_cast<unsigned int>(flags)));
    }
}
