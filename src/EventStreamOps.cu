#include <cuweaver/EventStreamOps.cuh>

#include "cuweaver_utils/ErrorCheck.cuh"

namespace cuweaver {
    float eventElapsedTime(const cudaEvent& start, const cudaEvent& end) {
        float ms;
        CUW_THROW_IF_ERROR(hipEventElapsedTime(&ms, start.nativeHandle(), end.nativeHandle()));
        return ms;
    }

    bool eventQuery(const cudaEvent& event) {
        try {
            CUW_THROW_IF_ERROR(hipEventQuery(event.nativeHandle()));
            return true;
        } catch (const hipError_t& e) {
            if (e.codeNative() == hipErrorNotReady) {
                return false;
            }
            throw;
        }
    }
}
