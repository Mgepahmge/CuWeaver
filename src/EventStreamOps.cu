#include <cuweaver/EventStreamOps.cuh>

#include "cuweaver_utils/ErrorCheck.cuh"

namespace cuweaver {
    float eventElapsedTime(const cudaEvent& start, const cudaEvent& end) {
        float ms;
        CUW_THROW_IF_ERROR(hipEventElapsedTime(&ms, start.nativeHandle(), end.nativeHandle()));
        return ms;
    }

    bool eventQuery(const cudaEvent& event) {
        try {
            CUW_THROW_IF_ERROR(hipEventQuery(event.nativeHandle()));
            return true;
        }
        catch (const hipError_t& e) {
            if (e.codeNative() == hipErrorNotReady) {
                return false;
            }
            throw;
        }
    }

    void eventRecord(const cudaEvent& event, const cudaStream& stream) {
        CUW_THROW_IF_ERROR(hipEventRecord(event.nativeHandle(), stream.nativeHandle()));
    }

    void eventRecordWithFlags(const cudaEvent& event, const cudaStream& stream, cudaEventRecordFlags flags) {
        CUW_THROW_IF_ERROR(
            hipEventRecordWithFlags(event.nativeHandle(), stream.nativeHandle(), static_cast<unsigned int>(flags)));
    }

    void eventSynchronize(const cudaEvent& event) {
        CUW_THROW_IF_ERROR(hipEventSynchronize(event.nativeHandle()));
    }

    bool streamQuery(const cudaStream& stream) {
        try {
            CUW_THROW_IF_ERROR(hipStreamQuery(stream.nativeHandle()));
            return true;
        }
        catch (const hipError_t& e) {
            if (e.codeNative() == hipErrorNotReady) {
                return false;
            }
            throw;
        }
    }

    void streamSynchronize(const cudaStream& stream) {
        CUW_THROW_IF_ERROR(hipStreamSynchronize(stream.nativeHandle()));
    }

    void streamAddCallback(const cudaStream& stream, hipStreamCallback_t callback, void* userData,
                           const unsigned int flags) {
        if (flags) {
            throw std::invalid_argument("Use streamAddCallback with flags set to 0.");
        }
        CUW_THROW_IF_ERROR(hipStreamAddCallback(stream.nativeHandle(), callback, userData, flags));
    }

    void streamWaitEvent(const cudaStream& stream, const cudaEvent& event, cudaEventWait flags) {
        CUW_THROW_IF_ERROR(
            hipStreamWaitEvent(stream.nativeHandle(), event.nativeHandle(), static_cast<unsigned int>(flags)));
    }
}
