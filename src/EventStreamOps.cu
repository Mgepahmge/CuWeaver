#include <cuweaver/EventStreamOps.cuh>

#include "cuweaver_utils/ErrorCheck.cuh"

namespace cuweaver {
    float eventElapsedTime(const cudaEvent& start, const cudaEvent& end) {
        float ms;
        CUW_THROW_IF_ERROR(hipEventElapsedTime(&ms, start.nativeHandle(), end.nativeHandle()));
        return ms;
    }

    bool eventQuery(const cudaEvent& event) {
        try {
            CUW_THROW_IF_ERROR(hipEventQuery(event.nativeHandle()));
            return true;
        }
        catch (const hipError_t& e) {
            if (e.codeNative() == hipErrorNotReady) {
                return false;
            }
            throw;
        }
    }

    void eventRecord(const cudaEvent& event, const cudaStream& stream) {
        CUW_THROW_IF_ERROR(hipEventRecord(event.nativeHandle(), stream.nativeHandle()));
    }

    void eventRecordWithFlags(const cudaEvent& event, const cudaStream& stream, cudaEventRecordFlags flags) {
        CUW_THROW_IF_ERROR(
            hipEventRecordWithFlags(event.nativeHandle(), stream.nativeHandle(), static_cast<unsigned int>(flags)));
    }

    void eventSynchronize(const cudaEvent& event) {
        CUW_THROW_IF_ERROR(hipEventSynchronize(event.nativeHandle()));
    }

    bool streamQuery(const cudaStream& stream) {
        try {
            CUW_THROW_IF_ERROR(hipStreamQuery(stream.nativeHandle()));
            return true;
        }
        catch (const hipError_t& e) {
            if (e.codeNative() == hipErrorNotReady) {
                return false;
            }
            throw;
        }
    }

    void streamSynchronize(const cudaStream& stream) {
        CUW_THROW_IF_ERROR(hipStreamSynchronize(stream.nativeHandle()));
    }

    void streamAddCallback(const cudaStream& stream, hipStreamCallback_t callback, void* userData,
                           const unsigned int flags) {
        if (flags) {
            throw std::invalid_argument("Use streamAddCallback with flags set to 0.");
        }
        CUW_THROW_IF_ERROR(hipStreamAddCallback(stream.nativeHandle(), callback, userData, flags));
    }

    void streamWaitEvent(const cudaStream& stream, const cudaEvent& event, cudaEventWait flags) {
        CUW_THROW_IF_ERROR(
            hipStreamWaitEvent(stream.nativeHandle(), event.nativeHandle(), static_cast<unsigned int>(flags)));
    }

    /**
     * @brief Launches a host function to execute on a CUDA stream.
     *
     * @details Submits a host-side function `fn` to the underlying CUDA stream of `stream`. The host function will execute
     *          after all preceding operations in the stream have completed. The `userData` pointer is passed to `fn` as its
     *          argument when invoked. This function wraps `hipLaunchHostFunc` and throws an exception on failure.
     *
     * @param[in] stream cudaStream wrapper whose underlying stream will schedule the host function.
     * @param[in] fn Host function to execute (must adhere to the `hipHostFn_t` signature: `void (*)(void*)`).
     * @param[in] userData User-defined data pointer passed to `fn` during execution.
     *
     * @throws hipError_t Thrown if `hipLaunchHostFunc` fails (e.g., invalid stream handle, null function pointer, or CUDA runtime error).
     *
     * @par Returns
     *      Nothing.
     */
    void launchHostFunc(const cudaStream& stream, hipHostFn_t fn, void* userData) {
        CUW_THROW_IF_ERROR(hipLaunchHostFunc(stream.nativeHandle(), fn, userData));
    }
}
